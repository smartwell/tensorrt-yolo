#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include "yololayer.h"

using namespace YoloLayer;

hipError_t forwardGpu(const float *const * inputs,float * output, hipStream_t stream,int batchSize,std::vector<YoloLayer::YoloKernel> yolokernel, int mThreadCount);

__device__ float Logist(float data){ return 1.0f / (1.0f + expf(-data)); };

__global__ void CalDetection(const float *input, float *output,int noElements,
                             int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes,int outputElem) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= noElements) return;

    int total_grid = yoloWidth * yoloHeight;
    int bnIdx = idx / total_grid;
    idx = idx - total_grid*bnIdx;
    int info_len_i = 5 + classes;
    const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

    for (int k = 0; k < 3; ++k) {
        int class_id = 0;
        float max_cls_prob = 0.0;
        for (int i = 5; i < info_len_i; ++i) {
            float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
            if (p > max_cls_prob) {
                max_cls_prob = p;
                class_id = i - 5;
            }
        }
        float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
        if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH) continue;

        float *res_count = output + bnIdx*outputElem;
        int count = (int)atomicAdd(res_count, 1);
        if (count >= MAX_OUTPUT_BBOX_COUNT) return;
        char* data = (char * )res_count + sizeof(float) + count*sizeof(Detection);
        Detection* det =  (Detection*)(data);

        int row = idx / yoloWidth;
        int col = idx % yoloWidth;

        //Location
        det->bbox[0] = (col + Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * INPUT_W / yoloWidth;
        det->bbox[1] = (row + Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * INPUT_H / yoloHeight;
        det->bbox[2] = expf(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]) * anchors[2*k];
        det->bbox[3] = expf(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]) * anchors[2*k + 1];
        det->det_confidence = box_prob;
        det->class_id = class_id;
        det->class_confidence = max_cls_prob;
    }
}

hipError_t forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize,std::vector<YoloLayer::YoloKernel> mYoloKernel, int mThreadCount) {
    void* devAnchor;
    size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
    hipMalloc(&devAnchor,AnchorLen);

    int outputElem = 1 + MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

    for(int idx = 0 ; idx < batchSize; ++idx) {
        hipMemset(output + idx*outputElem, 0, sizeof(float));
    }
    int numElem = 0;
    for (unsigned int i = 0;i< mYoloKernel.size();++i)
    {
        const auto& yolo = mYoloKernel[i];
        numElem = yolo.width*yolo.height*batchSize;
        if (numElem < mThreadCount)
            mThreadCount = numElem;
        hipMemcpy(devAnchor, yolo.anchors, AnchorLen, hipMemcpyHostToDevice);
        CalDetection<<< (yolo.width * yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount>>>
        (inputs[i], output, numElem, yolo.width, yolo.height, (float *)devAnchor, 80, outputElem);
    }

    hipFree(devAnchor);
    return hipGetLastError();
}



